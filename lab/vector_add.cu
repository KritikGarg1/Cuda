
#include <hip/hip_runtime.h>
#include <stdio.h>
//using namespace std;
//#typedef n 100

// Kernel Definition

__global__ void VecAddKernel(float *d_A, float *d_B, float *d_C, int n){
  int i=blockDim.x*blockIdx.x+threadIdx.x;
  if(i<n) d_C[i]=d_A[i]+d_B[i];
}
void vecAdd(float *A, float *B, float *C, int n){
  float *d_A, *d_B, *d_C;
  int size=n*sizeof(float);

  // Device Memory Allocation
  hipMalloc((void**)&d_A, size);
  hipMalloc((void**)&d_B, size);
  hipMalloc((void**)&d_C, size);

  // Host to Device data transfer
  hipMemcpy(d_A,A,size,hipMemcpyHostToDevice);
  hipMemcpy(d_B,B,size,hipMemcpyHostToDevice);
  
  // Calling Kernel
  VecAddKernel<<< ceil(n/16),16>>> (d_A,d_B,d_C,n);
  hipMemcpy(C,d_C,size,hipMemcpyDeviceToHost);

}


int main(){
  
  float *A, *B, *C;
  int n;
  
  printf("Enter the size of Vector");
  scanf("%d",&n);
  
  A = (float*)malloc(n*sizeof(float));
  B = (float*)malloc(n*sizeof(float));
  C = (float*)malloc(n*sizeof(float));

  for(int i=0;i<n;i++){
    A[i]=i;
    B[i]=i*i;
  }
  
  vecAdd(A,B,C,n);
  printf("The value of A+B .i.e  C = \n{");
  
  for(int i=0;i<n;i++){
    printf("%f, ",C[i]);
  }
  
  printf("}\n");

  return 0;
}
