#include "hip/hip_runtime.h"
// Rishabh Agarwal - 18JE0676
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
using namespace std;

// kernel function

__global__ void kernelFunction(int *a, int *b, int *c, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        int idx1 = (idx + 1) % 256;
        int idx2 = (idx + 2) % 256;
        float   as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
        float   bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
        c[idx] = (as + bs)/2;
    }
}


int main( void ) {

    hipDeviceProp_t  prop;
    int whichDevice;
    hipGetDevice(&whichDevice);
    hipGetDeviceProperties(&prop, whichDevice);
    
    if (!prop.deviceOverlap) {
        cout << "Device will not handle overlaps, so no speed up from streams\n";
        return 0;
    }    
    if(prop.concurrentKernels == 0) {
        cout << "> GPU does not support concurrent kernel execution\n";
        cout << "  CUDA kernel runs will be serialized\n";
    }
    if(prop.asyncEngineCount == 0) {
        cout << "GPU does not support concurrent Data transer and overlaping of kernel execution & data transfer\n";
        cout << "Mem copy call will be blocking calls\n";
    }

    hipEvent_t start, stop;
    float elapsedTime;

    int n = 1024*1024;
    int maxsize = n*20;

    int *ha, *hb, *hc;
    int *da0, *db0, *dc0, *da1, *db1, *dc1;
    hipStream_t    stream0, stream1;

    // start the timers
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // initialize the streams
    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);

    // allocate the memory on the GPU
    hipMalloc(&da0, n * sizeof(int));
    hipMalloc(&da1, n * sizeof(int));
    hipMalloc(&db0, n * sizeof(int));
    hipMalloc(&db1, n * sizeof(int));
    hipMalloc(&dc0, n * sizeof(int));
    hipMalloc(&dc1, n * sizeof(int));

    // allocate host locked memory, used to stream
    hipHostAlloc((void**)&ha, maxsize * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&hb, maxsize * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&hc, maxsize * sizeof(int), hipHostMallocDefault);

    for(int i=0; i < maxsize; i++) {
        ha[i] = i + 10;
        hb[i] = i + 200;
    }

    hipEventRecord(start, 0);
    for(int i=0; i < maxsize; i += n*2) {
        
        // enqueue copies of a in stream0 and stream1
        hipMemcpyAsync(da0, ha + i, n * sizeof(int), hipMemcpyHostToDevice, stream0);
        hipMemcpyAsync(da1, ha + i + n, n * sizeof(int), hipMemcpyHostToDevice, stream1);
        
        // enqueue copies of b in stream0 and stream1
        hipMemcpyAsync(db0, hb + i, n * sizeof(int), hipMemcpyHostToDevice, stream0);
        hipMemcpyAsync(db1, hb + i + n, n * sizeof(int), hipMemcpyHostToDevice, stream1);

        // enqueue kernels in stream0 and stream1   
        kernelFunction <<< n/256, 256, 0, stream0 >>> (da0, db0, dc0, n);
        kernelFunction <<< n/256, 256, 0, stream1 >>> (da1, db1, dc1, n);

        // enqueue copies of c from device to locked memory
        hipMemcpyAsync(hc + i, dc0, n * sizeof(int), hipMemcpyDeviceToHost, stream0);
        hipMemcpyAsync(hc + i + n, dc1, n * sizeof(int), hipMemcpyDeviceToHost, stream1);
    }

    hipStreamSynchronize(stream0);
    hipStreamSynchronize(stream1);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    cout << "Time taken in ms: " <<  elapsedTime << "\n\n";


    // we are printing only upto 20 elements
    cout << "Vector A: \n";
    for(int i=0; i < 20; i++) {
        cout << ha[i] << " ";
    }
    cout << "\n\n";

    cout << "Vector B: \n";
    for(int i=0; i < 20; i++) {
        cout << hb[i] << " ";
    }
    cout << "\n\n";

    cout <<"After performing operation: C[i] = ((A[i] + A[i+1] + A[i+2]) / 3 + (B[i] + B[i+1] + B[i+2]) / 3) / 2\n";
    cout << "Vector C: \n";
    for(int i=0; i < 20; i++) {
        cout << hc[i] << " ";
    }
    cout << "\n\n";

    hipHostFree(ha);
    hipHostFree(hb);
    hipHostFree(hc);
    
    hipFree(da0);
    hipFree(da1);
    hipFree(db0);
    hipFree(db1);
    hipFree(dc0);
    hipFree(dc1);
    
    hipStreamDestroy(stream0);
    hipStreamDestroy(stream1);

    return 0;
}
